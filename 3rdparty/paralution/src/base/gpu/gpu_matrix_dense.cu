#include "hip/hip_runtime.h"
// *************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2012-2013 Dimitar Lukarski
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// *************************************************************************

#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_matrix_dia.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_matrix_hyb.hpp"
#include "gpu_matrix_mcsr.hpp"
#include "gpu_matrix_bcsr.hpp"
#include "gpu_matrix_dense.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_dense.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_dense.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <assert.h>
#include <hip/hip_runtime.h>


namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::GPUAcceleratorMatrixDENSE() {

  // no default constructors
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::GPUAcceleratorMatrixDENSE(const Paralution_Backend_Descriptor local_backend) {

  this->mat_.val = NULL;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  FATAL_ERROR(__FILE__, __LINE__);
}


template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::~GPUAcceleratorMatrixDENSE() {

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixDENSE<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::AllocateDENSE(const int nrow, const int ncol) {

  assert( ncol  >= 0);
  assert( nrow  >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nrow*ncol > 0) {

    allocate_gpu(nrow*ncol, &this->mat_.val);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nrow*ncol, mat_.val);   

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nrow*ncol;

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.val);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert((this->get_nnz()  == src.get_nnz())  &&
	   (this->get_nrow() == src.get_nrow()) &&
	   (this->get_ncol() == src.get_ncol()) );

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixDENSE<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateDENSE(this->get_nrow(), this->get_ncol() );

    assert((this->get_nnz()  == dst->get_nnz())  &&
	   (this->get_nrow() == dst->get_nrow()) &&
	   (this->get_ncol() == dst->get_ncol()) );

    if (this->get_nnz() > 0) {
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert((this->get_nnz()  == src.get_nnz())  &&
	   (this->get_nrow() == src.get_nrow()) &&
	   (this->get_ncol() == src.get_ncol()) );

    if (this->get_nnz() > 0) { 

        hipMemcpy(this->mat_.val,         // dst
                   gpu_cast_mat->mat_.val, // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixDENSE<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateDENSE(dst->get_nrow(), dst->get_ncol() );

    assert((this->get_nnz()  == dst->get_nnz())  &&
	   (this->get_nrow() == dst->get_nrow()) &&
	   (this->get_ncol() == dst->get_ncol()) );

    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixDENSE<ValueType>   *cast_mat_dense;
  
  if ((cast_mat_dense = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_dense);
      return true;

  }

  /*
  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;
    
  }
  */

  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {
/*
    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in) ; 
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);
*/
    FATAL_ERROR(__FILE__, __LINE__);    
  }

}


template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {
  FATAL_ERROR(__FILE__, __LINE__);
}


template class GPUAcceleratorMatrixDENSE<double>;
template class GPUAcceleratorMatrixDENSE<float>;

}

