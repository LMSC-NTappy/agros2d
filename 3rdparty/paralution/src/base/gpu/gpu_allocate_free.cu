// *************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2012-2013 Dimitar Lukarski
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// *************************************************************************

#include <cmath>
#include "gpu_allocate_free.hpp"
#include <hip/hip_runtime.h>
#include <assert.h>
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"

namespace paralution {

template <typename DataType>
void allocate_gpu(const int size, DataType **ptr) {

  if (size > 0) {

    assert(*ptr == NULL);
    
    hipMalloc( (void **)ptr, size*sizeof(DataType));
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    assert(*ptr != NULL);
  }

};

template <typename DataType>
void free_gpu(DataType **ptr) {

  assert(*ptr != NULL);
  
  hipDeviceSynchronize();
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  hipFree(*ptr);
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  
  *ptr = NULL;

};

template <typename DataType>
void set_to_zero_gpu(const int blocksize,
                     const int max_threads,
                     const int size, DataType *ptr) {
  
  if (size > 0) {

    assert(ptr != NULL);

    hipMemset(ptr, 0, size*sizeof(DataType));
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    /*
    int s = size ;
    int k = (size/blocksize)/max_threads + 1;       
    if (k > 1) s = size / k;
    

    dim3 BlockSize(blocksize);
    dim3 GridSize(s / blocksize + 1);

    hipDeviceSynchronize();
    
    kernel_set_to_zeros<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
    */

    /*
    // 1D accessing, no stride
    dim3 BlockSize(blocksize);
    dim3 GridSize(size / blocksize + 1);

    hipDeviceSynchronize();
    
    kernel_set_to_zeros<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
    */
    
  }

}


template <typename DataType>
void set_to_one_gpu(const int blocksize,
                    const int max_threads,
                    const int size, DataType *ptr) {
  
  if (size > 0) {

    assert(ptr != NULL);


    int s = size ;
    int k = (size/blocksize)/max_threads + 1;       
    if (k > 1) s = size / k;
    

    dim3 BlockSize(blocksize);
    dim3 GridSize(s / blocksize + 1);

    hipDeviceSynchronize();
    
    kernel_set_to_ones<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      

    /*
    // 1D accessing, no stride
    dim3 BlockSize(blocksize);
    dim3 GridSize(size / blocksize + 1);

    hipDeviceSynchronize();
    
    kernel_set_to_ones<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
    */

  }

}



template void allocate_gpu<float       >(const int size, float        **ptr);
template void allocate_gpu<double      >(const int size, double       **ptr);
template void allocate_gpu<int         >(const int size, int          **ptr);
template void allocate_gpu<unsigned int>(const int size, unsigned int **ptr);
template void allocate_gpu<char        >(const int size, char         **ptr);

template void free_gpu<float       >(float        **ptr);
template void free_gpu<double      >(double       **ptr);
template void free_gpu<int         >(int          **ptr);
template void free_gpu<unsigned int>(unsigned int **ptr);
template void free_gpu<char        >(char         **ptr);

template void set_to_zero_gpu<float       >(const int blocksize, const int max_threads, const int size, float        *ptr);
template void set_to_zero_gpu<double      >(const int blocksize, const int max_threads, const int size, double       *ptr);
template void set_to_zero_gpu<int         >(const int blocksize, const int max_threads, const int size, int          *ptr);
template void set_to_zero_gpu<unsigned int>(const int blocksize, const int max_threads, const int size, unsigned int *ptr);
template void set_to_zero_gpu<char        >(const int blocksize, const int max_threads, const int size, char         *ptr);


template void set_to_one_gpu<float       >(const int blocksize, const int max_threads, const int size, float        *ptr);
template void set_to_one_gpu<double      >(const int blocksize, const int max_threads, const int size, double       *ptr);
template void set_to_one_gpu<int         >(const int blocksize, const int max_threads, const int size, int          *ptr);
template void set_to_one_gpu<unsigned int>(const int blocksize, const int max_threads, const int size, unsigned int *ptr);
template void set_to_one_gpu<char        >(const int blocksize, const int max_threads, const int size, char         *ptr);


};


