#include "hip/hip_runtime.h"
// *************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2012-2013 Dimitar Lukarski
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// *************************************************************************

#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_matrix_dia.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_matrix_hyb.hpp"
#include "gpu_matrix_mcsr.hpp"
#include "gpu_matrix_bcsr.hpp"
#include "gpu_matrix_dense.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_dia.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "../../utils/allocate_free.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_dia.hpp"
#include "cuda_kernels_vector.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>


namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixDIA<ValueType>::GPUAcceleratorMatrixDIA() {

  // no default constructors
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixDIA<ValueType>::GPUAcceleratorMatrixDIA(const Paralution_Backend_Descriptor local_backend) {

  this->mat_.val = NULL;
  this->mat_.offset = NULL;  
  this->mat_.num_diag = 0 ;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixDIA<ValueType>::~GPUAcceleratorMatrixDIA() {

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixDIA<ValueType> diag=" << this->get_ndiag() << " nnz=" << this->get_nnz() );

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::AllocateDIA(const int nnz, const int nrow, const int ncol, const int ndiag) {

  assert(nnz >= 0);
  assert(ncol >= 0);
  assert(nrow >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    assert(ndiag > 0);


    allocate_gpu(nnz, &this->mat_.val);
    allocate_gpu(ndiag, &this->mat_.offset);
 
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, mat_.val);
    
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    ndiag, mat_.offset);

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nnz;
    this->mat_.num_diag = ndiag;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.val);
    free_gpu(&this->mat_.offset);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;
    this->mat_.num_diag = 0 ;

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixDIA<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixDIA<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDIA(cast_mat->get_nnz(), cast_mat->get_nrow(), cast_mat->get_ncol(), cast_mat->get_ndiag());

    assert((this->get_nnz()  == src.get_nnz())  &&
	   (this->get_nrow() == src.get_nrow()) &&
	   (this->get_ncol() == src.get_ncol()) );

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.offset,     // dst
                 cast_mat->mat_.offset, // src
                 this->get_ndiag()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    }
      
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixDIA<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixDIA<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateDIA(this->get_nnz(), this->get_nrow(), this->get_ncol(), this->get_ndiag());

    assert((this->get_nnz()  == dst->get_nnz())  &&
	   (this->get_nrow() == dst->get_nrow()) &&
	   (this->get_ncol() == dst->get_ncol()) );

    if (this->get_nnz() > 0) {

      hipMemcpy(cast_mat->mat_.offset, // dst
                 this->mat_.offset,     // src
                 this->get_ndiag()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixDIA<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixDIA<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDIA(gpu_cast_mat->get_nnz(), gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol(), gpu_cast_mat->get_ndiag());

    assert((this->get_nnz()  == src.get_nnz())  &&
	   (this->get_nrow() == src.get_nrow()) &&
	   (this->get_ncol() == src.get_ncol()) );

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.offset,         // dst
                 gpu_cast_mat->mat_.offset, // src
                 this->get_ndiag()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
               this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixDIA<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixDIA<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateDIA(gpu_cast_mat->get_nnz(), gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol(), gpu_cast_mat->get_ndiag());

    assert((this->get_nnz()  == dst->get_nnz())  &&
	   (this->get_nrow() == dst->get_nrow()) &&
	   (this->get_ncol() == dst->get_ncol()) );

    if (this->get_nnz() > 0) { 

      hipMemcpy(gpu_cast_mat->mat_.offset, // dst
                 this->mat_.offset,         // src
                 this->get_ndiag()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.val, // dst
                 this->mat_.val,         // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixDIA<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixDIA<ValueType>   *cast_mat_dia;
  
  if ((cast_mat_dia = dynamic_cast<const GPUAcceleratorMatrixDIA<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_dia);
      return true;

  }

  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {

    this->Clear();

    int nrow = cast_mat_csr->get_nrow();
    int ncol = cast_mat_csr->get_ncol();
    int *diag_map = NULL;

    // Get diagonal mapping vector
    allocate_gpu<int>(nrow+ncol, &diag_map);

    set_to_zero_gpu(this->local_backend_.GPU_block_size,
                    this->local_backend_.GPU_max_threads,
                    nrow+ncol, diag_map);

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    hipDeviceSynchronize();

    kernel_dia_diag_map<int> <<<GridSize, BlockSize>>> (nrow, cast_mat_csr->mat_.row_offset,
                                                        cast_mat_csr->mat_.col, diag_map);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    // Reduction to obtain number of occupied diagonals
    int *d_buffer = NULL;
    int *h_buffer = NULL;
    int GROUP_SIZE;
    int LOCAL_SIZE;
    int FinalReduceSize;

    allocate_gpu<int>(this->local_backend_.GPU_wrap * 4, &d_buffer);

    dim3 GridSize2(this->local_backend_.GPU_wrap * 4);

    GROUP_SIZE = ( size_t( ( size_t( nrow+ncol / ( this->local_backend_.GPU_wrap * 4 ) ) + 1 ) 
                 / this->local_backend_.GPU_block_size ) + 1 ) * this->local_backend_.GPU_block_size;
    LOCAL_SIZE = GROUP_SIZE / this->local_backend_.GPU_block_size;

    hipDeviceSynchronize();

    kernel_reduce<int, int, 256> <<<GridSize2, BlockSize>>> (nrow+ncol, diag_map, d_buffer, GROUP_SIZE, LOCAL_SIZE);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    FinalReduceSize = this->local_backend_.GPU_wrap * 4;
    allocate_host(FinalReduceSize, &h_buffer);

    hipMemcpy(h_buffer, // dst
               d_buffer, // src
               FinalReduceSize*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu<int>(&d_buffer);

    int num_diag = 0;
    for ( int i=0; i<FinalReduceSize; ++i )
      num_diag += h_buffer[i];

    free_host(&h_buffer);

    int nnz_dia;
    if (nrow < ncol)
      nnz_dia = ncol * num_diag;
    else
      nnz_dia = nrow * num_diag;

    // Allocate DIA structure
    this->AllocateDIA(nnz_dia, nrow, ncol, num_diag);

    set_to_zero_gpu(this->local_backend_.GPU_block_size,
                    this->local_backend_.GPU_max_threads,
                    nnz_dia, this->mat_.val);
    set_to_zero_gpu(this->local_backend_.GPU_block_size,
                    this->local_backend_.GPU_max_threads,
                    num_diag, this->mat_.offset);

    // Fill diagonal offset array
    allocate_gpu<int>(nrow+ncol+1, &d_buffer);

    // TODO currently performing partial sum on host
    allocate_host(nrow+ncol+1, &h_buffer);
    hipMemcpy(h_buffer+1, // dst
               diag_map, // src
               (nrow+ncol)*sizeof(int), // size
               hipMemcpyDeviceToHost);

    h_buffer[0] = 0;
    for (int i=2; i<nrow+ncol+1; ++i)
      h_buffer[i] += h_buffer[i-1];

    hipMemcpy(d_buffer, // dst
               h_buffer, // src
               (nrow+ncol)*sizeof(int), // size
               hipMemcpyHostToDevice);

    free_host(&h_buffer);
    // end TODO

    dim3 GridSize3((nrow+ncol) / this->local_backend_.GPU_block_size + 1);

    hipDeviceSynchronize();

    kernel_dia_fill_offset<int> <<<GridSize3, BlockSize>>> (nrow, ncol, diag_map,
                                                            d_buffer, this->mat_.offset);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);
    free_gpu<int>(&d_buffer);

    hipDeviceSynchronize();

    kernel_dia_convert<ValueType, int> <<<GridSize, BlockSize>>> (nrow, num_diag, cast_mat_csr->mat_.row_offset,
                                                                  cast_mat_csr->mat_.col, cast_mat_csr->mat_.val,
                                                                  diag_map, this->mat_.val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);
    free_gpu<int>(&diag_map);

    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = nnz_dia;
    this->mat_.num_diag = num_diag;

    return true;

  }

  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in) ; 
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out) ; 

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    int nrow = this->get_nrow();
    int ncol = this->get_ncol();
    int num_diag = this->get_ndiag();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    hipDeviceSynchronize();
    kernel_dia_spmv<ValueType, int> <<<GridSize, BlockSize>>> (nrow, ncol, num_diag,
                                                               this->mat_.offset, this->mat_.val,
                                                               cast_in->vec_, cast_out->vec_ );
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}


template <typename ValueType>
void GPUAcceleratorMatrixDIA<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in) ; 
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out) ; 

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    int nrow = this->get_nrow();
    int ncol = this->get_ncol();
    int num_diag = this->get_ndiag();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    hipDeviceSynchronize();
    kernel_dia_add_spmv<ValueType, int> <<<GridSize, BlockSize>>> (nrow, ncol, num_diag,
                                                                   this->mat_.offset, this->mat_.val,
                                                                   scalar,
                                                                   cast_in->vec_, cast_out->vec_ );
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}


template class GPUAcceleratorMatrixDIA<double>;
template class GPUAcceleratorMatrixDIA<float>;

}

